#include "hip/hip_runtime.h"
// Includes CUDA
#include <hip/hip_runtime.h>
#include <cstdio>
#include <hipcub/hipcub.hpp>

const int vectorSize = 25;
const int vectorPerBlock = 1024;

__global__ void distanceKernel(unsigned int* keyOutput, unsigned int *valueOutput, unsigned int *query, hipTextureObject_t texObj, unsigned int texHeight)
{
    int tu = blockDim.x * blockIdx.x;
    int tv = threadIdx.x;

    if (tu <texHeight && tv <vectorPerBlock)
    {
        __shared__ unsigned int queryLocal[vectorSize];

        if (tv < vectorSize)
        {
            queryLocal[tv] = query[tv];
        }

        __syncthreads();

        unsigned int count = 0;

        for (int i = 0; i<vectorSize;++i)
        {
            unsigned int m = tex2D<unsigned int>(texObj, tv * vectorSize + i, tu);
            count += __popc(m ^ queryLocal[i]);

        }

        unsigned int id = tu*vectorPerBlock + tv;

        keyOutput[id] = count;
        valueOutput[id] = id;
    }
}

__host__ bool kNN(unsigned int *matrixBuffer, const unsigned int num_items, unsigned int query[vectorSize], unsigned int *result, const unsigned int resultCount)
{
    int texHeight = (num_items / vectorPerBlock) + ((num_items % vectorPerBlock)?1:0);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0 , 0, 0, hipChannelFormatKindUnsigned);
    hipArray *matrixArray_device = 0;
    hipError_t error = hipSuccess;
    bool success = false;
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    hipTextureObject_t matrixTexObj = 0;
    unsigned int *query_device = 0;
    unsigned int *keyResultArray_device = 0;
    unsigned int *valueResultArray_device = 0;
    size_t  tempStorageBytes  = 0;
    void    *tempStorage_device     = NULL;
    hipcub::CachingDeviceAllocator  cubAllocator(true);
    hipcub::DoubleBuffer<unsigned int> keys_device;
    hipcub::DoubleBuffer<unsigned int> values_device;

    error = hipMallocArray(&matrixArray_device, &channelDesc, vectorSize * vectorPerBlock, texHeight);
    if (hipSuccess != error)
    {
        printf("can't allocate matrixArray [%u]\n", error);
        goto cleanup;
    }

    error = hipMemcpyToArray(matrixArray_device, 0, 0, matrixBuffer, num_items * vectorSize * sizeof(unsigned int), hipMemcpyHostToDevice);
    if (hipSuccess != error)
    {
        printf("can't memcpy matrixArray [%u]\n", error);
        goto cleanup;
    }

    // Specify texture
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = matrixArray_device;

    // Specify texture object parameters
    texDesc.addressMode[0] = hipAddressModeWrap;
    texDesc.addressMode[1] = hipAddressModeWrap;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    // Create texture object
    error = hipCreateTextureObject(&matrixTexObj, &resDesc, &texDesc, NULL);
    if (hipSuccess != error)
    {
        printf("can't allocate texture object [%u]\n", error);
        goto cleanup;
    }

    error = hipMalloc(&query_device, sizeof(unsigned int) * vectorSize);
    if (hipSuccess != error)
    {
        printf("can't allocate for query\n");
        goto cleanup;
    }

    error = hipMemcpy(query_device, query, sizeof(unsigned int) * vectorSize, hipMemcpyHostToDevice);
    if (hipSuccess != error)
    {
        printf("can't memcpy query\n");
        goto cleanup;
    }

    error = hipMalloc(&keyResultArray_device, sizeof(unsigned int) * num_items);
    if (hipSuccess != error)
    {
        printf("can't allocate for key\n");
        goto cleanup;
    }

    error = hipMalloc(&valueResultArray_device, sizeof(unsigned int) * num_items);
    if (hipSuccess != error)
    {
        printf("can't allocate for value\n");
        goto cleanup;
    }

    distanceKernel<<<texHeight, vectorPerBlock>>>(keyResultArray_device, valueResultArray_device, query_device, matrixTexObj, texHeight);

    /* debug code {
        unsigned int *debugKey = new unsigned int [2000];

        hipMemcpy(debugKey, keyResultArray_device, 2000*sizeof(unsigned int), hipMemcpyDeviceToHost);

        printf("debug: %d, %d, %d\n", debugKey[1025], debugKey[500], debugKey[502]);
    }*/

    keys_device.d_buffers[keys_device.selector] = keyResultArray_device;
    values_device.d_buffers[values_device.selector] = valueResultArray_device;
    CubDebugExit(cubAllocator.DeviceAllocate((void**)&keys_device.d_buffers[keys_device.selector ^ 1], sizeof(unsigned int) * num_items));
    CubDebugExit(cubAllocator.DeviceAllocate((void**)&values_device.d_buffers[values_device.selector ^ 1], sizeof(unsigned int) * num_items));
    // Allocate temporary storage
    CubDebugExit(hipcub::DeviceRadixSort::SortPairs(tempStorage_device, tempStorageBytes, keys_device, values_device, num_items));
    CubDebugExit(cubAllocator.DeviceAllocate(&tempStorage_device, tempStorageBytes));

    // Run
    CubDebugExit(hipcub::DeviceRadixSort::SortPairs(tempStorage_device, tempStorageBytes, keys_device, values_device, num_items));

    hipMemcpy(result, values_device.Current(), sizeof(unsigned int) * resultCount, hipMemcpyDeviceToHost);

    success = true;
cleanup:
    return true;
    if (matrixTexObj)
    {
        hipDestroyTextureObject(matrixTexObj);
    }

    if (matrixArray_device)
    {
        hipFreeArray(matrixArray_device);
    }

    if (query_device)
    {
        hipFree(query_device);
    }

    if (keys_device.d_buffers[0] || keys_device.d_buffers[1])
    {
        hipFree(keys_device.d_buffers[0]);
        hipFree(keys_device.d_buffers[1]);
    }
    else if(keyResultArray_device)
    {
        hipFree(keyResultArray_device);
    }

    if (values_device.d_buffers[0] || values_device.d_buffers[1])
    {
        hipFree(values_device.d_buffers[0]);
        hipFree(values_device.d_buffers[1]);
    }
    else if(valueResultArray_device)
    {
        hipFree(valueResultArray_device);
    }

    if (tempStorage_device)
    {
        hipFree(tempStorage_device);
    }

    return success;
}
